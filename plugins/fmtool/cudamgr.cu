
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

extern "C" {

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

int open_cuda_dev(int cudadev) {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  printf("Detected %d CUDA accelerators:\n", deviceCount);
  int dev;
  for (dev=0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("  CUDA device[%d]: '%s'  Mem: %dMB  Rev: %d.%d\n",
           dev, deviceProp.name, deviceProp.totalGlobalMem / (1024*1024),
           deviceProp.major, deviceProp.minor);
  }

  if (cudadev < 0 || cudadev >= deviceCount) {
    printf("No such CUDA device %d, using device 0\n", cudadev);
    cudadev = 0;
  }
  hipSetDevice(cudadev);
  CUERR // check and clear any existing errors

  return 0;
}

}
