#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007-2011 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAQuickSurf.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.81 $      $Date: 2016/04/20 04:57:46 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *   CUDA accelerated gaussian density calculation
 *
 ***************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#if CUDART_VERSION < 4000
#error The VMD QuickSurf feature requires CUDA 4.0 or later
#endif

#include "Inform.h"
#include "utilities.h"
#include "WKFThreads.h"
#include "WKFUtils.h"
#include "CUDAKernels.h" 
#include "CUDASpatialSearch.h"
#include "CUDAMarchingCubes.h"
#include "CUDAQuickSurf.h" 

#include "DispCmds.h"
#include "VMDDisplayList.h"

#if 1
#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
  printf("Thread aborting...\n"); \
  return NULL; }}
#else
#define CUERR
#endif


//
// density format conversion routines
//

// no-op conversion for float to float
inline __device__ void convert_density(float & df, float df2) {
  df = df2;
}

// Convert float (32-bit) to half-precision (16-bit floating point) stored
// into an unsigned short (16-bit integer type). 
inline __device__ void convert_density(unsigned short & dh, float df2) {
  dh = __float2half_rn(df2);
}



//
// color format conversion routines
//

// No-op conversion for float3 to float3
inline __device__ void convert_color(float3 & cf, float3 cf2) {
  cf = cf2;
}

// Convert float3 colors to uchar4, performing the necessary bias, scaling, 
// and range clamping so we don't encounter integer wraparound, etc.
inline __device__ void convert_color(uchar4 & cu, float3 cf) {
  // conversion to GLubyte format, Table 2.6, p. 44 of OpenGL spec 1.2.1
  // c = f * (2^8-1)

  // scale color values to prevent overflow, 
  // and convert to fixed-point representation all at once
  float invmaxcolscale = __frcp_rn(fmaxf(fmaxf(fmaxf(cf.x, cf.y), cf.z), 1.0f)) * 255.0f;

  // clamp color values to prevent integer wraparound
  cu = make_uchar4(cf.x * invmaxcolscale,
                   cf.y * invmaxcolscale,
                   cf.z * invmaxcolscale,
                   255);
}

// convert uchar4 colors to float3
inline __device__ void convert_color(float3 & cf, uchar4 cu) {
  const float i2f = 1.0f / 255.0f;
  cf.x = cu.x * i2f;
  cf.y = cu.y * i2f;
  cf.z = cu.z * i2f;
}


//
// Restrict macro to make it easy to do perf tuning tests
//
#if 0
#define RESTRICT __restrict__
#else
#define RESTRICT
#endif

// 
// Parameters for linear-time range-limited gaussian density kernels
//
#define GGRIDSZ   8.0f
#define GBLOCKSZX 8
#define GBLOCKSZY 8

#if 1
#define GTEXBLOCKSZZ 2
#define GTEXUNROLL   4
#define GBLOCKSZZ    2
#define GUNROLL      4
#else
#define GTEXBLOCKSZZ 8
#define GTEXUNROLL   1
#define GBLOCKSZZ    8
#define GUNROLL      1
#endif

#if __CUDA_ARCH__ >= 300
#define MAXTHRDENS  ( GBLOCKSZX * GBLOCKSZY * GBLOCKSZZ )
#define MINBLOCKDENS 1
#elif __CUDA_ARCH__ >= 200
#define MAXTHRDENS  ( GBLOCKSZX * GBLOCKSZY * GBLOCKSZZ )
#define MINBLOCKDENS 1
#else
#define MAXTHRDENS  ( GBLOCKSZX * GBLOCKSZY * GBLOCKSZZ )
#define MINBLOCKDENS 1
#endif


//
// Templated version of the density map kernel to handle multiple 
// data formats for the output density volume and volumetric texture.
// This variant of the density map algorithm normalizes densities so
// that the target isovalue is a density of 1.0.
//
template<class DENSITY, class VOLTEX>
__global__ static void 
// __launch_bounds__ ( MAXTHRDENS, MINBLOCKDENS )
gaussdensity_fast_tex_norm(int natoms,
                      const float4 * RESTRICT sorted_xyzr, 
                      const float4 * RESTRICT sorted_color, 
                      int3 volsz,
                      int3 acncells,
                      float acgridspacing,
                      float invacgridspacing,
                      const uint2 * RESTRICT cellStartEnd,
                      float gridspacing, unsigned int z, 
                      DENSITY * RESTRICT densitygrid,
                      VOLTEX * RESTRICT voltexmap,
                      float invisovalue) {
  unsigned int xindex  = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int yindex  = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int zindex  = ((blockIdx.z * blockDim.z) + threadIdx.z) * GTEXUNROLL;

  // shave register use slightly
  unsigned int outaddr = zindex * volsz.x * volsz.y + 
                         yindex * volsz.x + xindex;

  // early exit if this thread is outside of the grid bounds
  if (xindex >= volsz.x || yindex >= volsz.y || zindex >= volsz.z)
    return;

  zindex += z;

  // compute ac grid index of lower corner minus gaussian radius
  int xabmin = ((blockIdx.x * blockDim.x) * gridspacing - acgridspacing) * invacgridspacing;
  int yabmin = ((blockIdx.y * blockDim.y) * gridspacing - acgridspacing) * invacgridspacing;
  int zabmin = ((z + blockIdx.z * blockDim.z * GTEXUNROLL) * gridspacing - acgridspacing) * invacgridspacing;

  // compute ac grid index of upper corner plus gaussian radius
  int xabmax = (((blockIdx.x+1) * blockDim.x) * gridspacing + acgridspacing) * invacgridspacing;
  int yabmax = (((blockIdx.y+1) * blockDim.y) * gridspacing + acgridspacing) * invacgridspacing;
  int zabmax = ((z + (blockIdx.z+1) * blockDim.z * GTEXUNROLL) * gridspacing + acgridspacing) * invacgridspacing;

  xabmin = (xabmin < 0) ? 0 : xabmin;
  yabmin = (yabmin < 0) ? 0 : yabmin;
  zabmin = (zabmin < 0) ? 0 : zabmin;
  xabmax = (xabmax >= acncells.x-1) ? acncells.x-1 : xabmax;
  yabmax = (yabmax >= acncells.y-1) ? acncells.y-1 : yabmax;
  zabmax = (zabmax >= acncells.z-1) ? acncells.z-1 : zabmax;

  float coorx = gridspacing * xindex;
  float coory = gridspacing * yindex;
  float coorz = gridspacing * zindex;

  float densityval1=0.0f;
  float3 densitycol1=make_float3(0.0f, 0.0f, 0.0f);
#if GTEXUNROLL >= 2
  float densityval2=0.0f;
  float3 densitycol2=densitycol1;
#endif
#if GTEXUNROLL >= 4
  float densityval3=0.0f;
  float3 densitycol3=densitycol1;
  float densityval4=0.0f;
  float3 densitycol4=densitycol1;
#endif

  int acplanesz = acncells.x * acncells.y;
  int xab, yab, zab;
  for (zab=zabmin; zab<=zabmax; zab++) {
    for (yab=yabmin; yab<=yabmax; yab++) {
      for (xab=xabmin; xab<=xabmax; xab++) {
        int abcellidx = zab * acplanesz + yab * acncells.x + xab;
        uint2 atomstartend = cellStartEnd[abcellidx];
        if (atomstartend.x != GRID_CELL_EMPTY) {
          unsigned int atomid;
          for (atomid=atomstartend.x; atomid<atomstartend.y; atomid++) {
            float4 atom  = sorted_xyzr[atomid];
            float4 color = sorted_color[atomid];
            float dx = coorx - atom.x;
            float dy = coory - atom.y;
            float dxy2 = dx*dx + dy*dy;

            float dz = coorz - atom.z;
            float r21 = (dxy2 + dz*dz) * atom.w;
            float tmp1 = invisovalue * exp2f(r21); // normalized density
            densityval1 += tmp1;
            densitycol1.x += tmp1 * color.x;
            densitycol1.y += tmp1 * color.y;
            densitycol1.z += tmp1 * color.z;

#if GTEXUNROLL >= 2
            float dz2 = dz + gridspacing;
            float r22 = (dxy2 + dz2*dz2) * atom.w;
            float tmp2 = invisovalue * exp2f(r22); // normalized density
            densityval2 += tmp2;
            densitycol2.x += tmp2 * color.x;
            densitycol2.y += tmp2 * color.y;
            densitycol2.z += tmp2 * color.z;
#endif
#if GTEXUNROLL >= 4
            float dz3 = dz2 + gridspacing;
            float r23 = (dxy2 + dz3*dz3) * atom.w;
            float tmp3 = invisovalue * exp2f(r23); // normalized density
            densityval3 += tmp3;
            densitycol3.x += tmp3 * color.x;
            densitycol3.y += tmp3 * color.y;
            densitycol3.z += tmp3 * color.z;

            float dz4 = dz3 + gridspacing;
            float r24 = (dxy2 + dz4*dz4) * atom.w;
            float tmp4 = invisovalue * exp2f(r24); // normalized density
            densityval4 += tmp4;
            densitycol4.x += tmp4 * color.x;
            densitycol4.y += tmp4 * color.y;
            densitycol4.z += tmp4 * color.z;
#endif
          }
        }
      }
    }
  }

  DENSITY densityout;
  VOLTEX texout;
  convert_density(densityout, densityval1);
  densitygrid[outaddr          ] = densityout;
  convert_color(texout, densitycol1);
  voltexmap[outaddr          ] = texout;

#if GTEXUNROLL >= 2
  int planesz = volsz.x * volsz.y;
  convert_density(densityout, densityval2);
  densitygrid[outaddr + planesz] = densityout;
  convert_color(texout, densitycol2);
  voltexmap[outaddr + planesz] = texout;
#endif
#if GTEXUNROLL >= 4
  convert_density(densityout, densityval3);
  densitygrid[outaddr + 2*planesz] = densityout;
  convert_color(texout, densitycol3);
  voltexmap[outaddr + 2*planesz] = texout;

  convert_density(densityout, densityval4);
  densitygrid[outaddr + 3*planesz] = densityout;
  convert_color(texout, densitycol4);
  voltexmap[outaddr + 3*planesz] = texout;
#endif
}


__global__ static void 
// __launch_bounds__ ( MAXTHRDENS, MINBLOCKDENS )
gaussdensity_fast_tex3f(int natoms,
                        const float4 * RESTRICT sorted_xyzr, 
                        const float4 * RESTRICT sorted_color, 
                        int3 volsz,
                        int3 acncells,
                        float acgridspacing,
                        float invacgridspacing,
                        const uint2 * RESTRICT cellStartEnd,
                        float gridspacing, unsigned int z, 
                        float * RESTRICT densitygrid,
                        float3 * RESTRICT voltexmap,
                        float invisovalue) {
  unsigned int xindex  = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int yindex  = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int zindex  = ((blockIdx.z * blockDim.z) + threadIdx.z) * GTEXUNROLL;

  // shave register use slightly
  unsigned int outaddr = zindex * volsz.x * volsz.y + 
                         yindex * volsz.x + xindex;

  // early exit if this thread is outside of the grid bounds
  if (xindex >= volsz.x || yindex >= volsz.y || zindex >= volsz.z)
    return;

  zindex += z;

  // compute ac grid index of lower corner minus gaussian radius
  int xabmin = ((blockIdx.x * blockDim.x) * gridspacing - acgridspacing) * invacgridspacing;
  int yabmin = ((blockIdx.y * blockDim.y) * gridspacing - acgridspacing) * invacgridspacing;
  int zabmin = ((z + blockIdx.z * blockDim.z * GTEXUNROLL) * gridspacing - acgridspacing) * invacgridspacing;

  // compute ac grid index of upper corner plus gaussian radius
  int xabmax = (((blockIdx.x+1) * blockDim.x) * gridspacing + acgridspacing) * invacgridspacing;
  int yabmax = (((blockIdx.y+1) * blockDim.y) * gridspacing + acgridspacing) * invacgridspacing;
  int zabmax = ((z + (blockIdx.z+1) * blockDim.z * GTEXUNROLL) * gridspacing + acgridspacing) * invacgridspacing;

  xabmin = (xabmin < 0) ? 0 : xabmin;
  yabmin = (yabmin < 0) ? 0 : yabmin;
  zabmin = (zabmin < 0) ? 0 : zabmin;
  xabmax = (xabmax >= acncells.x-1) ? acncells.x-1 : xabmax;
  yabmax = (yabmax >= acncells.y-1) ? acncells.y-1 : yabmax;
  zabmax = (zabmax >= acncells.z-1) ? acncells.z-1 : zabmax;

  float coorx = gridspacing * xindex;
  float coory = gridspacing * yindex;
  float coorz = gridspacing * zindex;

  float densityval1=0.0f;
  float3 densitycol1=make_float3(0.0f, 0.0f, 0.0f);
#if GTEXUNROLL >= 2
  float densityval2=0.0f;
  float3 densitycol2=densitycol1;
#endif
#if GTEXUNROLL >= 4
  float densityval3=0.0f;
  float3 densitycol3=densitycol1;
  float densityval4=0.0f;
  float3 densitycol4=densitycol1;
#endif

  int acplanesz = acncells.x * acncells.y;
  int xab, yab, zab;
  for (zab=zabmin; zab<=zabmax; zab++) {
    for (yab=yabmin; yab<=yabmax; yab++) {
      for (xab=xabmin; xab<=xabmax; xab++) {
        int abcellidx = zab * acplanesz + yab * acncells.x + xab;
        uint2 atomstartend = cellStartEnd[abcellidx];
        if (atomstartend.x != GRID_CELL_EMPTY) {
          unsigned int atomid;
          for (atomid=atomstartend.x; atomid<atomstartend.y; atomid++) {
            float4 atom  = sorted_xyzr[atomid];
            float4 color = sorted_color[atomid];
            float dx = coorx - atom.x;
            float dy = coory - atom.y;
            float dxy2 = dx*dx + dy*dy;

            float dz = coorz - atom.z;
            float r21 = (dxy2 + dz*dz) * atom.w;
            float tmp1 = exp2f(r21);
            densityval1 += tmp1;
            tmp1 *= invisovalue;
            densitycol1.x += tmp1 * color.x;
            densitycol1.y += tmp1 * color.y;
            densitycol1.z += tmp1 * color.z;

#if GTEXUNROLL >= 2
            float dz2 = dz + gridspacing;
            float r22 = (dxy2 + dz2*dz2) * atom.w;
            float tmp2 = exp2f(r22);
            densityval2 += tmp2;
            tmp2 *= invisovalue;
            densitycol2.x += tmp2 * color.x;
            densitycol2.y += tmp2 * color.y;
            densitycol2.z += tmp2 * color.z;
#endif
#if GTEXUNROLL >= 4
            float dz3 = dz2 + gridspacing;
            float r23 = (dxy2 + dz3*dz3) * atom.w;
            float tmp3 = exp2f(r23);
            densityval3 += tmp3;
            tmp3 *= invisovalue;
            densitycol3.x += tmp3 * color.x;
            densitycol3.y += tmp3 * color.y;
            densitycol3.z += tmp3 * color.z;

            float dz4 = dz3 + gridspacing;
            float r24 = (dxy2 + dz4*dz4) * atom.w;
            float tmp4 = exp2f(r24);
            densityval4 += tmp4;
            tmp4 *= invisovalue;
            densitycol4.x += tmp4 * color.x;
            densitycol4.y += tmp4 * color.y;
            densitycol4.z += tmp4 * color.z;
#endif
          }
        }
      }
    }
  }

  densitygrid[outaddr          ] = densityval1;
  voltexmap[outaddr          ].x = densitycol1.x;
  voltexmap[outaddr          ].y = densitycol1.y;
  voltexmap[outaddr          ].z = densitycol1.z;

#if GTEXUNROLL >= 2
  int planesz = volsz.x * volsz.y;
  densitygrid[outaddr + planesz] = densityval2;
  voltexmap[outaddr + planesz].x = densitycol2.x;
  voltexmap[outaddr + planesz].y = densitycol2.y;
  voltexmap[outaddr + planesz].z = densitycol2.z;
#endif
#if GTEXUNROLL >= 4
  densitygrid[outaddr + 2*planesz] = densityval3;
  voltexmap[outaddr + 2*planesz].x = densitycol3.x;
  voltexmap[outaddr + 2*planesz].y = densitycol3.y;
  voltexmap[outaddr + 2*planesz].z = densitycol3.z;

  densitygrid[outaddr + 3*planesz] = densityval4;
  voltexmap[outaddr + 3*planesz].x = densitycol4.x;
  voltexmap[outaddr + 3*planesz].y = densitycol4.y;
  voltexmap[outaddr + 3*planesz].z = densitycol4.z;
#endif
}


__global__ static void 
// __launch_bounds__ ( MAXTHRDENS, MINBLOCKDENS )
gaussdensity_fast(int natoms,
                  const float4 * RESTRICT sorted_xyzr, 
                  int3 volsz,
                  int3 acncells,
                  float acgridspacing,
                  float invacgridspacing,
                  const uint2 * RESTRICT cellStartEnd,
                  float gridspacing, unsigned int z, 
                  float * RESTRICT densitygrid) {
  unsigned int xindex  = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int yindex  = (blockIdx.y * blockDim.y) + threadIdx.y;
  unsigned int zindex  = ((blockIdx.z * blockDim.z) + threadIdx.z) * GUNROLL;
  unsigned int outaddr = zindex * volsz.x * volsz.y + 
                         yindex * volsz.x + 
                         xindex;

  // early exit if this thread is outside of the grid bounds
  if (xindex >= volsz.x || yindex >= volsz.y || zindex >= volsz.z)
    return;

  zindex += z;

  // compute ac grid index of lower corner minus gaussian radius
  int xabmin = ((blockIdx.x * blockDim.x) * gridspacing - acgridspacing) * invacgridspacing;
  int yabmin = ((blockIdx.y * blockDim.y) * gridspacing - acgridspacing) * invacgridspacing;
  int zabmin = ((z + blockIdx.z * blockDim.z * GUNROLL) * gridspacing - acgridspacing) * invacgridspacing;

  // compute ac grid index of upper corner plus gaussian radius
  int xabmax = (((blockIdx.x+1) * blockDim.x) * gridspacing + acgridspacing) * invacgridspacing;
  int yabmax = (((blockIdx.y+1) * blockDim.y) * gridspacing + acgridspacing) * invacgridspacing;
  int zabmax = ((z + (blockIdx.z+1) * blockDim.z * GUNROLL) * gridspacing + acgridspacing) * invacgridspacing;

  xabmin = (xabmin < 0) ? 0 : xabmin;
  yabmin = (yabmin < 0) ? 0 : yabmin;
  zabmin = (zabmin < 0) ? 0 : zabmin;
  xabmax = (xabmax >= acncells.x-1) ? acncells.x-1 : xabmax;
  yabmax = (yabmax >= acncells.y-1) ? acncells.y-1 : yabmax;
  zabmax = (zabmax >= acncells.z-1) ? acncells.z-1 : zabmax;

  float coorx = gridspacing * xindex;
  float coory = gridspacing * yindex;
  float coorz = gridspacing * zindex;

  float densityval1=0.0f;
#if GUNROLL >= 2
  float densityval2=0.0f;
#endif
#if GUNROLL >= 4
  float densityval3=0.0f;
  float densityval4=0.0f;
#endif

  int acplanesz = acncells.x * acncells.y;
  int xab, yab, zab;
  for (zab=zabmin; zab<=zabmax; zab++) {
    for (yab=yabmin; yab<=yabmax; yab++) {
      for (xab=xabmin; xab<=xabmax; xab++) {
        int abcellidx = zab * acplanesz + yab * acncells.x + xab;
        uint2 atomstartend = cellStartEnd[abcellidx];
        if (atomstartend.x != GRID_CELL_EMPTY) {
          unsigned int atomid;
          for (atomid=atomstartend.x; atomid<atomstartend.y; atomid++) {
            float4 atom = sorted_xyzr[atomid];
            float dx = coorx - atom.x;
            float dy = coory - atom.y;
            float dxy2 = dx*dx + dy*dy;
  
            float dz = coorz - atom.z;
            float r21 = (dxy2 + dz*dz) * atom.w;
            densityval1 += exp2f(r21);

#if GUNROLL >= 2
            float dz2 = dz + gridspacing;
            float r22 = (dxy2 + dz2*dz2) * atom.w;
            densityval2 += exp2f(r22);
#endif
#if GUNROLL >= 4
            float dz3 = dz2 + gridspacing;
            float r23 = (dxy2 + dz3*dz3) * atom.w;
            densityval3 += exp2f(r23);

            float dz4 = dz3 + gridspacing;
            float r24 = (dxy2 + dz4*dz4) * atom.w;
            densityval4 += exp2f(r24);
#endif
          }
        }
      }
    }
  }

  densitygrid[outaddr            ] = densityval1;
#if GUNROLL >= 2
  int planesz = volsz.x * volsz.y;
  densitygrid[outaddr +   planesz] = densityval2;
#endif
#if GUNROLL >= 4
  densitygrid[outaddr + 2*planesz] = densityval3;
  densitygrid[outaddr + 3*planesz] = densityval4;
#endif
}


// per-GPU handle with various memory buffer pointers, etc.
typedef struct {
  /// max grid sizes and attributes the current allocations will support
  int verbose;
  long int natoms;
  int colorperatom;
  int acx;
  int acy;
  int acz;
  int gx;
  int gy;
  int gz;

  CUDAMarchingCubes *mc;     ///< Marching cubes class used to extract surface

  float *devdensity;         ///< density map stored in GPU memory
  void *devvoltexmap;        ///< volumetric texture map
  float4 *xyzr_d;            ///< atom coords and radii
  float4 *sorted_xyzr_d;     ///< cell-sorted coords and radii
  float4 *color_d;           ///< colors
  float4 *sorted_color_d;    ///< cell-sorted colors

  unsigned int *atomIndex_d; ///< cell index for each atom
  unsigned int *atomHash_d;  ///<  
  uint2 *cellStartEnd_d;     ///< cell start/end indices 

  void *safety;
  float3 *v3f_d;
  float3 *n3f_d;
  float3 *c3f_d;
  char3 *n3b_d;
  uchar4 *c4u_d;
} qsurf_gpuhandle;


CUDAQuickSurf::CUDAQuickSurf() {
  voidgpu = calloc(1, sizeof(qsurf_gpuhandle));
  if (getenv("VMDQUICKSURFVERBOSE") != NULL) {
    qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;
    gpuh->verbose = 1;
    int tmp = atoi(getenv("VMDQUICKSURFVERBOSE"));
    if (tmp > 0)
      gpuh->verbose = tmp;
  }
}


CUDAQuickSurf::~CUDAQuickSurf() {
  qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;

  // free all working buffers if not done already
  free_bufs();

  // delete marching cubes object
  delete gpuh->mc;

  free(voidgpu);
}


int CUDAQuickSurf::free_bufs() {
  qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;

  // zero out max buffer capacities
  gpuh->natoms = 0;
  gpuh->colorperatom = 0;
  gpuh->acx = 0;
  gpuh->acy = 0;
  gpuh->acz = 0;
  gpuh->gx = 0;
  gpuh->gy = 0;
  gpuh->gz = 0;

  if (gpuh->safety != NULL)
    hipFree(gpuh->safety);
  gpuh->safety=NULL;

  if (gpuh->devdensity != NULL)
    hipFree(gpuh->devdensity);
  gpuh->devdensity=NULL;

  if (gpuh->devvoltexmap != NULL)
    hipFree(gpuh->devvoltexmap);
  gpuh->devvoltexmap=NULL;

  if (gpuh->xyzr_d != NULL)
    hipFree(gpuh->xyzr_d);
  gpuh->xyzr_d=NULL;

  if (gpuh->sorted_xyzr_d != NULL)
    hipFree(gpuh->sorted_xyzr_d);  
  gpuh->sorted_xyzr_d=NULL;

  if (gpuh->color_d != NULL)
    hipFree(gpuh->color_d);
  gpuh->color_d=NULL;

  if (gpuh->sorted_color_d != NULL)
    hipFree(gpuh->sorted_color_d);
  gpuh->sorted_color_d=NULL;

  if (gpuh->atomIndex_d != NULL)
    hipFree(gpuh->atomIndex_d);
  gpuh->atomIndex_d=NULL;

  if (gpuh->atomHash_d != NULL)
    hipFree(gpuh->atomHash_d);
  gpuh->atomHash_d=NULL;

  if (gpuh->cellStartEnd_d != NULL)
    hipFree(gpuh->cellStartEnd_d);
  gpuh->cellStartEnd_d=NULL;

  if (gpuh->v3f_d != NULL)
    hipFree(gpuh->v3f_d);
  gpuh->v3f_d=NULL;

  if (gpuh->n3f_d != NULL)
    hipFree(gpuh->n3f_d);
  gpuh->n3f_d=NULL;

  if (gpuh->c3f_d != NULL)
    hipFree(gpuh->c3f_d);
  gpuh->c3f_d=NULL;

  if (gpuh->n3b_d != NULL)
    hipFree(gpuh->n3b_d);
  gpuh->n3b_d=NULL;

  if (gpuh->c4u_d != NULL)
    hipFree(gpuh->c4u_d);
  gpuh->c4u_d=NULL;


  return 0;
}


int CUDAQuickSurf::check_bufs(long int natoms, int colorperatom, 
                              int acx, int acy, int acz,
                              int gx, int gy, int gz) {
  qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;

  // If the current atom count, texturing mode, and total voxel count
  // use the same or less storage than the size of the existing buffers,
  // we can reuse the same buffers without having to go through the 
  // complex allocation and validation loops.  This is a big performance
  // benefit during trajectory animation.
  if (natoms <= gpuh->natoms &&
      colorperatom <= gpuh->colorperatom &&
      (acx*acy*acz) <= (gpuh->acx * gpuh->acy * gpuh->acz) && 
      (gx*gy*gz) <= (gpuh->gx * gpuh->gy * gpuh->gz))
    return 0;
 
  return -1; // no existing bufs, or too small to be used 
}


int CUDAQuickSurf::alloc_bufs(long int natoms, int colorperatom, 
                              VolTexFormat vtexformat, 
                              int acx, int acy, int acz,
                              int gx, int gy, int gz) {
  qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;

  // early exit from allocation call if we've already got existing
  // buffers that are large enough to support the request
  if (check_bufs(natoms, colorperatom, acx, acy, acz, gx, gy, gz) == 0)
    return 0;

  // If we have any existing allocations, trash them as they weren't
  // usable for this new request and we need to reallocate them from scratch
  free_bufs();

  long int acncells = ((long) acx) * ((long) acy) * ((long) acz);
  long int ncells = ((long) gx) * ((long) gy) * ((long) gz);
  long int volmemsz = ncells * sizeof(float);
  long int chunkmaxverts = 3L * ncells; // assume worst case 50% triangle occupancy
  long int MCsz = CUDAMarchingCubes::MemUsageMC(gx, gy, gz);

  // Allocate all of the memory buffers our algorithms will need up-front,
  // so we can retry and gracefully reduce the sizes of various buffers
  // to attempt to fit within available GPU memory 
  long int totalmemsz = 
    volmemsz +                                       // volume
    (2L * natoms * sizeof(unsigned int)) +           // bin sort
    (acncells * sizeof(uint2)) +                     // bin sort
    (3L * chunkmaxverts * sizeof(float3)) +          // MC vertex bufs 
    natoms*sizeof(float4) +                          // thrust
    8L * gx * gy * sizeof(float) +                   // thrust
    MCsz;                                            // mcubes

  hipMalloc((void**)&gpuh->devdensity, volmemsz);
  if (colorperatom) {
    int voltexsz = 0;
    switch (vtexformat) {
      case RGB3F:
        voltexsz = ncells * sizeof(float3);
        break;

      case RGB4U:
        voltexsz = ncells * sizeof(uchar4);
        break;
    }
    hipMalloc((void**)&gpuh->devvoltexmap, voltexsz);
    hipMalloc((void**)&gpuh->color_d, natoms * sizeof(float4));
    hipMalloc((void**)&gpuh->sorted_color_d, natoms * sizeof(float4));
    totalmemsz += 2 * natoms * sizeof(float4);
  }
  hipMalloc((void**)&gpuh->xyzr_d, natoms * sizeof(float4));
  hipMalloc((void**)&gpuh->sorted_xyzr_d, natoms * sizeof(float4));
  hipMalloc((void**)&gpuh->atomIndex_d, natoms * sizeof(unsigned int));
  hipMalloc((void**)&gpuh->atomHash_d, natoms * sizeof(unsigned int));
  hipMalloc((void**)&gpuh->cellStartEnd_d, acncells * sizeof(uint2));

  // allocate marching cubes output buffers
  hipMalloc((void**)&gpuh->v3f_d, 3 * chunkmaxverts * sizeof(float3));
#if 1
  hipMalloc((void**)&gpuh->n3b_d, 3 * chunkmaxverts * sizeof(char3));
  totalmemsz += 3 * chunkmaxverts * sizeof(char3);   // MC normal bufs 
#else
  hipMalloc((void**)&gpuh->n3f_d, 3 * chunkmaxverts * sizeof(float3));
  totalmemsz += 3 * chunkmaxverts * sizeof(float3);  // MC normal bufs 
#endif
#if 1
  hipMalloc((void**)&gpuh->c4u_d, 3 * chunkmaxverts * sizeof(uchar4));
  totalmemsz += 3 * chunkmaxverts * sizeof(uchar4);  // MC vertex color bufs 
#else
  hipMalloc((void**)&gpuh->c3f_d, 3 * chunkmaxverts * sizeof(float3));
  totalmemsz += 3 * chunkmaxverts * sizeof(float3);  // MC vertex color bufs 
#endif

  // Allocate an extra phantom array to act as a safety net to
  // ensure that subsequent allocations performed internally by 
  // the NVIDIA thrust template library or by our 
  // marching cubes implementation don't fail, since we can't 
  // currently pre-allocate all of them.
  hipMalloc(&gpuh->safety, 
             natoms*sizeof(float4) +                          // thrust
             8 * gx * gy * sizeof(float) +                    // thrust
             MCsz);                                           // mcubes

  if (gpuh->verbose > 1)
    printf("Total QuickSurf mem size: %d MB\n", totalmemsz / (1024*1024));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return -1;

  // once the allocation has succeeded, we update the GPU handle info 
  // so that the next test/allocation pass knows the latest state.
  gpuh->natoms = natoms;
  gpuh->colorperatom = colorperatom;

  gpuh->acx = acx;
  gpuh->acy = acy;
  gpuh->acz = acz;

  gpuh->gx = gx;
  gpuh->gy = gy;
  gpuh->gz = gz;

  return 0;
}


int CUDAQuickSurf::get_chunk_bufs(int testexisting,
                                  long int natoms, int colorperatom, 
                                  VolTexFormat vtexformat,
                                  int acx, int acy, int acz,
                                  int gx, int gy, int gz,
                                  int &cx, int &cy, int &cz,
                                  int &sx, int &sy, int &sz) {
  dim3 Bsz(GBLOCKSZX, GBLOCKSZY, GBLOCKSZZ);
  if (colorperatom)
    Bsz.z = GTEXBLOCKSZZ;

  hipError_t err = hipGetLastError(); // eat error so next CUDA op succeeds

  // enter loop to attempt a single-pass computation, but if the
  // allocation fails, cut the chunk size Z dimension by half
  // repeatedly until we either run chunks of 8 planes at a time,
  // otherwise we assume it is hopeless.
  cz <<= 1; // premultiply by two to simplify loop body
  int chunkiters = 0;
  int chunkallocated = 0;
  while (!chunkallocated) {
    // Cut the Z chunk size in half
    chunkiters++;
    cz >>= 1;

    // if we've already dropped to a subvolume size, subtract off the
    // four extra Z planes from last time before we do the modulo padding
    // calculation so we don't hit an infinite loop trying to go below 
    // 16 planes due the padding math below.
    if (cz != gz)
      cz-=4;

    // Pad the chunk to a multiple of the computational tile size since
    // each thread computes multiple elements (unrolled in the Z direction)
    cz += (8 - (cz % 8));

    // The density map "slab" size is the chunk size but without the extra
    // plane used to copy the last plane of the previous chunk's density
    // into the start, for use by the marching cubes.
    sx = cx;
    sy = cy;
    sz = cz;

    // Add four extra Z-planes for copying the previous end planes into 
    // the start of the next chunk.
    cz+=4;

#if 0
    printf("  Trying slab size: %d (test: %d)\n", sz, testexisting);
#endif

#if 1
    // test to see if total number of thread blocks exceeds maximum
    // number we can reasonably run prior to a kernel timeout error
    dim3 tGsz((sx+Bsz.x-1) / Bsz.x, 
              (sy+Bsz.y-1) / Bsz.y,
              (sz+(Bsz.z*GUNROLL)-1) / (Bsz.z * GUNROLL));
    if (colorperatom) {
      tGsz.z = (sz+(Bsz.z*GTEXUNROLL)-1) / (Bsz.z * GTEXUNROLL);
    }
    if (tGsz.x * tGsz.y * tGsz.z > 65535)
      continue; 
#endif

    // Bail out if we can't get enough memory to run at least
    // 8 slices in a single pass (making sure we've freed any allocations
    // beforehand, so they aren't leaked).
    if (sz <= 8) {
      return -1;
    }
 
    if (testexisting) {
      if (check_bufs(natoms, colorperatom, acx, acy, acz, cx, cy, cz) != 0)
        continue;
    } else {
      if (alloc_bufs(natoms, colorperatom, vtexformat, acx, acy, acz, cx, cy, cz) != 0)
        continue;
    }

    chunkallocated=1;
  }

  return 0;
}


int CUDAQuickSurf::calc_surf(long int natoms, const float *xyzr_f, 
                             const float *colors_f,
                             int colorperatom,
                             float *origin, int *numvoxels, float maxrad,
                             float radscale, float gridspacing, 
                             float isovalue, float gausslim,
                             VMDDisplayList *cmdList) {
  qsurf_gpuhandle *gpuh = (qsurf_gpuhandle *) voidgpu;

  int vtexsize = 0;
  const VolTexFormat voltexformat = RGB4U; // XXX caller may want to set this
  switch (voltexformat) {
    case RGB3F: 
      vtexsize = sizeof(float3);
      break;

    case RGB4U: 
      vtexsize = sizeof(uchar4);
      break;
  }

  float4 *colors = (float4 *) colors_f;
  int3 volsz = make_int3(numvoxels[0], numvoxels[1], numvoxels[2]);

  int chunkmaxverts=0;
  int chunknumverts=0; 
  int numverts=0;
  int numfacets=0;

  wkf_timerhandle globaltimer = wkf_timer_create();
  wkf_timer_start(globaltimer);

  hipError_t err;
  hipDeviceProp_t deviceProp;
  int dev;
  if (hipGetDevice(&dev) != hipSuccess) {
    wkf_timer_destroy(globaltimer);
    return -1;
  }
 
  memset(&deviceProp, 0, sizeof(hipDeviceProp_t));
  
  if (hipGetDeviceProperties(&deviceProp, dev) != hipSuccess) {
    wkf_timer_destroy(globaltimer);
    err = hipGetLastError(); // eat error so next CUDA op succeeds
    return -1;
  }

  // This code currently requires compute capability 1.3 or 2.x
  // because we absolutely depend on hardware broadcasts for 
  // global memory reads by multiple threads reading the same element,
  // and the code more generally assumes the Fermi L1 cache and prefers
  // to launch 3-D grids where possible.  The current code will run on 
  // GT200 with reasonable performance so we allow it currently.  More
  // testing will be needed to determine if laptop integrated 
  // GT200 devices are truly fast enough to outrun quad core CPUs or
  // if we should trigger CPU fallback on devices with smaller SM counts.
  if ((deviceProp.major < 2) &&
      ((deviceProp.major == 1) && (deviceProp.minor < 3))) {
    wkf_timer_destroy(globaltimer);
    return -1;
  }

  // compute grid spacing for the acceleration grid
  float acgridspacing = gausslim * radscale * maxrad;

  // ensure acceleration grid spacing >= density grid spacing
  if (acgridspacing < gridspacing)
    acgridspacing = gridspacing;

  // Allocate output arrays for the gaussian density map and 3-D texture map
  // We test for errors carefully here since this is the most likely place
  // for a memory allocation failure due to the size of the grid.
  int3 chunksz = volsz;
  int3 slabsz = volsz;

  // An alternative scheme to minimize the QuickSurf GPU memory footprint
  if (getenv("VMDQUICKSURFMINMEM")) {
    if (volsz.z > 32) {
      slabsz.z = chunksz.z = 16;
    }
  }

  int3 accelcells;
  accelcells.x = max(int((volsz.x*gridspacing) / acgridspacing), 1);
  accelcells.y = max(int((volsz.y*gridspacing) / acgridspacing), 1);
  accelcells.z = max(int((volsz.z*gridspacing) / acgridspacing), 1);

  dim3 Bsz(GBLOCKSZX, GBLOCKSZY, GBLOCKSZZ);
  if (colorperatom)
    Bsz.z = GTEXBLOCKSZZ;

  // check to see if it's possible to use an existing allocation,
  // if so, just leave things as they are, and do the computation 
  // using the existing buffers
  if (gpuh->natoms == 0 ||
      get_chunk_bufs(1, natoms, colorperatom, voltexformat,
                     accelcells.x, accelcells.y, accelcells.z,
                     volsz.x, volsz.y, volsz.z,
                     chunksz.x, chunksz.y, chunksz.z,
                     slabsz.x, slabsz.y, slabsz.z) == -1) {
    // reset the chunksz and slabsz after failing to try and
    // fit them into the existing allocations...
    chunksz = volsz;
    slabsz = volsz;

    // reallocate the chunk buffers from scratch since we weren't
    // able to reuse them
    if (get_chunk_bufs(0, natoms, colorperatom, voltexformat,
                       accelcells.x, accelcells.y, accelcells.z,
                       volsz.x, volsz.y, volsz.z,
                       chunksz.x, chunksz.y, chunksz.z,
                       slabsz.x, slabsz.y, slabsz.z) == -1) {
      wkf_timer_destroy(globaltimer);
      return -1;
    }
  }
  chunkmaxverts = 3 * chunksz.x * chunksz.y * chunksz.z;

  // Free the "safety padding" memory we allocate to ensure we dont
  // have trouble with thrust calls that allocate their own memory later
  if (gpuh->safety != NULL)
    hipFree(gpuh->safety);
  gpuh->safety = NULL;

  if (gpuh->verbose > 1) {
    printf("  Using GPU chunk size: %d\n", chunksz.z);
    printf("  Accel grid(%d, %d, %d) spacing %f\n",
           accelcells.x, accelcells.y, accelcells.z, acgridspacing);
  }

  // pre-process the atom coordinates and radii as needed
  // short-term fix until a new CUDA kernel takes care of this
  int i, i4;
  float4 *xyzr = (float4 *) malloc(natoms * sizeof(float4));
  float log2e = log2(2.718281828);
  for (i=0,i4=0; i<natoms; i++,i4+=4) {
    xyzr[i].x = xyzr_f[i4    ];
    xyzr[i].y = xyzr_f[i4 + 1];
    xyzr[i].z = xyzr_f[i4 + 2];

    float scaledrad = xyzr_f[i4 + 3] * radscale;
    float arinv = -1.0f * log2e / (2.0f*scaledrad*scaledrad);

    xyzr[i].w = arinv;
  }
  hipMemcpy(gpuh->xyzr_d, xyzr, natoms * sizeof(float4), hipMemcpyHostToDevice);
  free(xyzr);

  if (colorperatom)
    hipMemcpy(gpuh->color_d, colors, natoms * sizeof(float4), hipMemcpyHostToDevice);
 
  // build uniform grid acceleration structure
  if (vmd_cuda_build_density_atom_grid(natoms, gpuh->xyzr_d, gpuh->color_d,
                                       gpuh->sorted_xyzr_d,
                                       gpuh->sorted_color_d,
                                       gpuh->atomIndex_d, gpuh->atomHash_d,
                                       gpuh->cellStartEnd_d, 
                                       accelcells, 1.0f / acgridspacing) != 0) {
    wkf_timer_destroy(globaltimer);
    free_bufs();
    return -1;
  }

  double sorttime = wkf_timer_timenow(globaltimer);
  double lastlooptime = sorttime;

  double densitykerneltime = 0.0f;
  double densitytime = 0.0f;
  double mckerneltime = 0.0f;
  double mctime = 0.0f; 
  double copycalltime = 0.0f;
  double copytime = 0.0f;

  float *volslab_d = NULL;
  void *texslab_d = NULL;

  int lzplane = GBLOCKSZZ * GUNROLL;
  if (colorperatom)
    lzplane = GTEXBLOCKSZZ * GTEXUNROLL;

  // initialize CUDA marching cubes class instance or rebuild it if needed
  uint3 mgsz = make_uint3(chunksz.x, chunksz.y, chunksz.z);
  if (gpuh->mc == NULL) {
    gpuh->mc = new CUDAMarchingCubes(); 
    if (!gpuh->mc->Initialize(mgsz)) {
      printf("QuickSurf call to MC Initialize() failed\n");
    }
  } else {
    uint3 mcmaxgridsize = gpuh->mc->GetMaxGridSize();
    if (slabsz.x > mcmaxgridsize.x ||
        slabsz.y > mcmaxgridsize.y ||
        slabsz.z > mcmaxgridsize.z) {
      if (gpuh->verbose)
        printf("*** QuickSurf Allocating new MC object...\n");
 
      // delete marching cubes object
      delete gpuh->mc;

      // create and initialize CUDA marching cubes class instance
      gpuh->mc = new CUDAMarchingCubes(); 

      if (!gpuh->mc->Initialize(mgsz)) {
        printf("QuickSurf MC Initialize() call failed to recreate MC object\n");
      }
    } 
  }

  int z;
  int chunkcount=0;
  float invacgridspacing = 1.0f / acgridspacing;
  float invisovalue = 1.0f / isovalue;
  for (z=0; z<volsz.z; z+=slabsz.z) {
    int3 curslab = slabsz;
    if (z+curslab.z > volsz.z)
      curslab.z = volsz.z - z; 
  
    int slabplanesz = curslab.x * curslab.y;

    dim3 Gsz((curslab.x+Bsz.x-1) / Bsz.x, 
             (curslab.y+Bsz.y-1) / Bsz.y,
             (curslab.z+(Bsz.z*GUNROLL)-1) / (Bsz.z * GUNROLL));
    if (colorperatom)
      Gsz.z = (curslab.z+(Bsz.z*GTEXUNROLL)-1) / (Bsz.z * GTEXUNROLL);

    // For SM 2.x, we can run the entire slab in one pass by launching
    // a 3-D grid of thread blocks.
    // If we are running on SM 1.x, we can only launch 1-D grids so we
    // must loop over planar grids until we have processed the whole slab.
    dim3 Gszslice = Gsz;
    if (deviceProp.major < 2)
      Gszslice.z = 1;

    if (gpuh->verbose > 1) {
      printf("CUDA device %d, grid size %dx%dx%d\n", 
             0, Gsz.x, Gsz.y, Gsz.z);
      printf("CUDA: vol(%d,%d,%d) accel(%d,%d,%d)\n",
             curslab.x, curslab.y, curslab.z,
             accelcells.x, accelcells.y, accelcells.z);
      printf("Z=%d, curslab.z=%d\n", z, curslab.z);
    }

    // For all but the first density slab, we copy the last four 
    // planes of the previous run into the start of the next run so
    // that we can extract the isosurface with no discontinuities
    if (z == 0) {
      volslab_d = gpuh->devdensity;
      if (colorperatom)
        texslab_d = gpuh->devvoltexmap;
    } else {
      int fourplanes = 4 * slabplanesz;
      hipMemcpy(gpuh->devdensity,
                 volslab_d + (slabsz.z-4) * slabplanesz, 
                 fourplanes * sizeof(float), hipMemcpyDeviceToDevice);
      volslab_d = gpuh->devdensity + fourplanes;

      if (colorperatom) {
        hipMemcpy(gpuh->devvoltexmap,
                   ((unsigned char *) texslab_d) + (slabsz.z-4) * slabplanesz * vtexsize, 
                   fourplanes * vtexsize, hipMemcpyDeviceToDevice);
        texslab_d = ((unsigned char *) gpuh->devvoltexmap) + fourplanes * vtexsize;
      }
    }

    // loop over the planes/slices in a slab and compute density and texture
    for (int lz=0; lz<Gsz.z; lz+=Gszslice.z) {
      int lzinc = lz * lzplane;
      float *volslice_d = volslab_d + lzinc * slabplanesz;

      if (colorperatom) {
        void *texslice_d = ((unsigned char *) texslab_d) + lzinc * slabplanesz * vtexsize;
        switch (voltexformat) {
          case RGB3F:
            gaussdensity_fast_tex3f<<<Gszslice, Bsz, 0>>>(natoms, 
                gpuh->sorted_xyzr_d, gpuh->sorted_color_d, 
                curslab, accelcells, acgridspacing, invacgridspacing, 
                gpuh->cellStartEnd_d, gridspacing, z+lzinc,
                volslice_d, (float3 *) texslice_d, invisovalue);
            break;

          case RGB4U:
            gaussdensity_fast_tex_norm<float, uchar4><<<Gszslice, Bsz, 0>>>(natoms, 
                gpuh->sorted_xyzr_d, gpuh->sorted_color_d, 
                curslab, accelcells, acgridspacing, invacgridspacing, 
                gpuh->cellStartEnd_d, gridspacing, z+lzinc,
                volslice_d, (uchar4 *) texslice_d, invisovalue);
            break;
        }
      } else {
        gaussdensity_fast<<<Gszslice, Bsz, 0>>>(natoms, gpuh->sorted_xyzr_d, 
            curslab, accelcells, acgridspacing, invacgridspacing, 
            gpuh->cellStartEnd_d, gridspacing, z+lzinc, volslice_d);
      }
    }
    hipDeviceSynchronize(); 
    densitykerneltime = wkf_timer_timenow(globaltimer);

#if 0
    printf("  CUDA mcubes..."); fflush(stdout);
#endif

    uint3 gvsz = make_uint3(curslab.x, curslab.y, curslab.z);

    // For all but the first density slab, we copy the last four
    // planes of the previous run into the start of the next run so
    // that we can extract the isosurface with no discontinuities
    if (z != 0)
      gvsz.z=curslab.z + 4;

    float3 bbox = make_float3(gvsz.x * gridspacing, gvsz.y * gridspacing,
                              gvsz.z * gridspacing);

    float3 gorigin = make_float3(origin[0], origin[1], 
                                 origin[2] + (z * gridspacing));
    if (z != 0)
      gorigin.z = origin[2] + ((z-4) * gridspacing);

#if 0
printf("\n  ... vsz: %d %d %d\n", gvsz.x, gvsz.y, gvsz.z);
printf("  ... org: %.2f %.2f %.2f\n", gorigin.x, gorigin.y, gorigin.z);
printf("  ... bxs: %.2f %.2f %.2f\n", bbox.x, bbox.y, bbox.y);
printf("  ... bbe: %.2f %.2f %.2f\n", gorigin.x+bbox.x, gorigin.y+bbox.y, gorigin.z+bbox.z);
#endif

    // If we are computing the volume using multiple passes, we have to 
    // overlap the marching cubes grids and compute a sub-volume to exclude
    // the end planes, except for the first and last sub-volume, in order to
    // get correct per-vertex normals at the edges of each sub-volume 
    int skipstartplane=0;
    int skipendplane=0;
    if (chunksz.z < volsz.z) {
      // on any but the first pass, we skip the first Z plane
      if (z != 0)
        skipstartplane=1;

      // on any but the last pass, we skip the last Z plane
      if (z+curslab.z < volsz.z)
        skipendplane=1;
    }

    //
    // Extract density map isosurface using marching cubes
    //

    // Choose the isovalue dependingon whether the desnity map 
    // contains normalized or un-normalized density values
    if (voltexformat == RGB4U) {
      // incoming densities are pre-normalized so that the target isovalue
      // is represented as a density of 1.0f
      gpuh->mc->SetIsovalue(1.0f);
    } else {
      gpuh->mc->SetIsovalue(isovalue);
    }

    int mcstat = 0;
    switch (voltexformat) {
      case RGB3F:
        mcstat = gpuh->mc->SetVolumeData(gpuh->devdensity, 
                                         (float3 *) gpuh->devvoltexmap,
                                         gvsz, gorigin, bbox, true);
        break;

      case RGB4U:
        mcstat = gpuh->mc->SetVolumeData(gpuh->devdensity, 
                                         (uchar4 *) gpuh->devvoltexmap,
                                         gvsz, gorigin, bbox, true);
        break;
    }
    if (!mcstat) {
      printf("QuickSurf call to MC SetVolumeData() failed\n");
    }

    // set the sub-volume starting/ending indices if needed
    if (skipstartplane || skipendplane) {
      uint3 volstart = make_uint3(0, 0, 0);
      uint3 volend = make_uint3(gvsz.x, gvsz.y, gvsz.z);

      if (skipstartplane)
        volstart.z = 2;

      if (skipendplane)
        volend.z = gvsz.z - 2;

      gpuh->mc->SetSubVolume(volstart, volend);
    }
    if (gpuh->n3b_d) {
      gpuh->mc->computeIsosurface(gpuh->v3f_d, gpuh->n3b_d, 
                                  gpuh->c4u_d, chunkmaxverts);
    } else if (gpuh->c4u_d) {
      gpuh->mc->computeIsosurface(gpuh->v3f_d, gpuh->n3f_d, 
                                  gpuh->c4u_d, chunkmaxverts);
    } else {
      gpuh->mc->computeIsosurface(gpuh->v3f_d, gpuh->n3f_d, 
                                  gpuh->c3f_d, chunkmaxverts);
    }
    chunknumverts = gpuh->mc->GetVertexCount();

#if 0
    printf("generated %d vertices, max vert limit: %d\n", chunknumverts, chunkmaxverts);
#endif
    if (chunknumverts == chunkmaxverts)
      printf("  *** QuickSurf exceeded marching cubes vertex limit (%d verts)\n", chunknumverts);

    hipDeviceSynchronize(); 
    mckerneltime = wkf_timer_timenow(globaltimer);

    // Create a triangle mesh
    if (chunknumverts > 0) {
      DispCmdTriMesh cmdTriMesh;
      if (colorperatom) {
        // emit triangle mesh with per-vertex colors
        if (gpuh->n3b_d) {
          cmdTriMesh.cuda_putdata((const float *) gpuh->v3f_d, 
                                  (const char *) gpuh->n3b_d, 
                                  (const unsigned char *) gpuh->c4u_d,
                                  chunknumverts/3, cmdList);
        } else if (gpuh->c4u_d) {
          cmdTriMesh.cuda_putdata((const float *) gpuh->v3f_d, 
                                  (const float *) gpuh->n3f_d, 
                                  (const unsigned char *) gpuh->c4u_d,
                                  chunknumverts/3, cmdList);
        } else {
          cmdTriMesh.cuda_putdata((const float *) gpuh->v3f_d, 
                                  (const float *) gpuh->n3f_d, 
                                  (const float *) gpuh->c3f_d,
                                  chunknumverts/3, cmdList);
        }
      } else {
        // emit triangle mesh with no colors, uses current rendering state
        if (gpuh->n3b_d) {
          cmdTriMesh.cuda_putdata((const float *) gpuh->v3f_d, 
                                  (const char *) gpuh->n3b_d, 
                                  (const unsigned char *) NULL,
                                  chunknumverts/3, cmdList);
        } else {
          cmdTriMesh.cuda_putdata((const float *) gpuh->v3f_d,
                                  (const float *) gpuh->n3f_d, 
                                  (const float *) NULL,
                                  chunknumverts/3, cmdList);
        }
      }
    }
    numverts+=chunknumverts;
    numfacets+=chunknumverts/3;

#if 0
   // XXX we'll hold onto this as we'll want to rescue this approach
   //     for electrostatics coloring where we have to have the 
   //     entire triangle mesh in order to do the calculation
    int l;
    int vertstart = 3 * numverts;
    int vertbufsz = 3 * (numverts + chunknumverts) * sizeof(float);
    int facebufsz = (numverts + chunknumverts) * sizeof(int);
    int chunkvertsz = 3 * chunknumverts * sizeof(float);

    v = (float*) realloc(v, vertbufsz);
    n = (float*) realloc(n, vertbufsz);
    c = (float*) realloc(c, vertbufsz);
    f = (int*)   realloc(f, facebufsz);
    hipMemcpy(v+vertstart, gpuh->v3f_d, chunkvertsz, hipMemcpyDeviceToHost);
    hipMemcpy(n+vertstart, gpuh->n3f_d, chunkvertsz, hipMemcpyDeviceToHost);
    if (colorperatom) {
      hipMemcpy(c+vertstart, gpuh->c3f_d, chunkvertsz, hipMemcpyDeviceToHost);
    } else {
      float *color = c+vertstart;
      for (l=0; l<chunknumverts*3; l+=3) {
        color[l + 0] = colors[0].x;
        color[l + 1] = colors[0].y;
        color[l + 2] = colors[0].z;
      }
    }
    for (l=numverts; l<numverts+chunknumverts; l++) {
      f[l]=l;
    }
    numverts+=chunknumverts;
    numfacets+=chunknumverts/3;
#endif

    copycalltime = wkf_timer_timenow(globaltimer);

    densitytime += densitykerneltime - lastlooptime;
    mctime += mckerneltime - densitykerneltime;
    copytime += copycalltime - mckerneltime;

    lastlooptime = wkf_timer_timenow(globaltimer);

    chunkcount++; // increment number of chunks processed
  }

  // catch any errors that may have occured so that at the very least,
  // all of the subsequent resource deallocation calls will succeed
  err = hipGetLastError();

  wkf_timer_stop(globaltimer);
  double totalruntime = wkf_timer_time(globaltimer);
  wkf_timer_destroy(globaltimer);

  // If an error occured, we print it and return an error code, once
  // all of the memory deallocations have completed.
  if (err != hipSuccess) { 
    printf("CUDA error: %s, %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    return -1;
  }

  if (gpuh->verbose) {
    printf("  GPU generated %d vertices, %d facets, in %d passes\n", 
           numverts, numfacets, chunkcount);
    printf("  GPU time (%s): %.3f [sort: %.3f density %.3f mcubes: %.3f copy: %.3f]\n", 
           (deviceProp.major == 1 && deviceProp.minor == 3) ? "SM 1.3" : "SM 2.x",
           totalruntime, sorttime, densitytime, mctime, copytime);
  }

  return 0;
}





