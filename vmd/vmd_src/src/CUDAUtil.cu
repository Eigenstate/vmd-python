#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 1995-2016 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAUtil.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.41 $        $Date: 2016/11/28 03:04:58 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *   CUDA API wrapper for use by the CUDAAccel C++ class 
 ***************************************************************************/
#include <string.h>
#include <stdio.h>
#include "CUDAKernels.h"
#include "WKFThreads.h"

#if defined(__cplusplus)
extern "C" {
#endif

// report true if the driver is compatible with the runtime
static int vmd_cuda_drv_runtime_compatible() {
#if CUDART_VERSION >= 2020
  int cuda_driver_version=-1;
  int cuda_runtime_version=0;

  hipDriverGetVersion(&cuda_driver_version);
  hipRuntimeGetVersion(&cuda_runtime_version);

#if 0
  printf("CUDA driver version: %d\n", cuda_driver_version);
  printf("CUDA runtime version: %d\n", cuda_runtime_version);
#endif

  if (cuda_driver_version == 0) 
    return VMDCUDA_ERR_NODEVICES;

  if (cuda_driver_version < cuda_runtime_version) {
#if defined(ARCH_LINUXCARMA)
    // XXX workaround for the first native CUDA compiler toolchain (5.5)
    //     having a newer rev than the driver (310.32, CUDA 5.0) reports
    if (cuda_driver_version == 5000 && cuda_runtime_version == 5050)
      return VMDCUDA_ERR_NONE;
#endif
    return VMDCUDA_ERR_DRVMISMATCH;
  }
#endif  

  return VMDCUDA_ERR_NONE;
}


int vmd_cuda_device_props(int dev, char *name, int namelen,
                          int *devmajor, int *devminor, 
                          unsigned long *memb, int *clockratekhz,
                          int *smcount, int *integratedgpu,
                          int *asyncenginecount, int *kerneltimeout,
                          int *canmaphostmem, int *computemode) {
  hipError_t rc;
  hipDeviceProp_t deviceProp;

  int vercheck;
  if ((vercheck = vmd_cuda_drv_runtime_compatible()) != VMDCUDA_ERR_NONE) {
    return vercheck;
  }

  memset(&deviceProp, 0, sizeof(hipDeviceProp_t));
  if ((rc=hipGetDeviceProperties(&deviceProp, dev)) != hipSuccess) {
    // printf("error: %s\n", hipGetErrorString(rc));
    if (rc == cudaErrorNotYetImplemented)
      return VMDCUDA_ERR_EMUDEVICE;
    return VMDCUDA_ERR_GENERAL;
  }

  if (name)
    strncpy(name, deviceProp.name, namelen);
  if (devmajor)
    *devmajor = deviceProp.major;
  if (devminor)
    *devminor = deviceProp.minor;
  if (memb)
    *memb = deviceProp.totalGlobalMem;
  if (clockratekhz)
    *clockratekhz = deviceProp.clockRate;
#if CUDART_VERSION >= 2000
  if (smcount)
    *smcount = deviceProp.multiProcessorCount;
#else
  if (smcount)
    *smcount = -1;
#endif
#if CUDART_VERSION >= 4000
  if (asyncenginecount)
    *asyncenginecount = deviceProp.asyncEngineCount;
#elif CUDART_VERSION >= 2000
  // deviceProp.deviceOverlap is deprecated now...
  if (asyncenginecount)
    *asyncenginecount = (deviceProp.deviceOverlap != 0);
#else
  if (asyncenginecount)
    *asyncenginecount = 0; 
#endif
#if CUDART_VERSION >= 2010
  if (kerneltimeout)
    *kerneltimeout = (deviceProp.kernelExecTimeoutEnabled != 0);
#else
  if (kerneltimeout)
    *kerneltimeout = 0;
#endif
#if CUDART_VERSION >= 2020
  if (integratedgpu)
    *integratedgpu = (deviceProp.integrated != 0);
  if (canmaphostmem)
    *canmaphostmem = (deviceProp.canMapHostMemory != 0);
  if (computemode)
    *computemode = deviceProp.computeMode;
#else
  if (integratedgpu)
    *integratedgpu = 0;
  if (canmaphostmem)
    *canmaphostmem = 0;
  if (computemode)
    *computemode = VMDCUDA_COMPUTEMODE_DEFAULT;
#endif
  return VMDCUDA_ERR_NONE;
}


int vmd_cuda_num_devices(int *numdev) {
  int i;
  int devcount=0;
  int usabledevs=0;
  *numdev = 0;

  int vercheck;
  if ((vercheck = vmd_cuda_drv_runtime_compatible()) != VMDCUDA_ERR_NONE) {
    return vercheck;
  }

  if (hipGetDeviceCount(&devcount) != hipSuccess) {
    return VMDCUDA_ERR_NODEVICES;
  }

  // Do a sanity check in case we get complete gibberish back,
  // but no error. This can occur if we have either a driver or 
  // CUDA runtime that's badly mismatched.
  if (devcount > 100 || devcount < 0)
    return VMDCUDA_ERR_DRVMISMATCH;

  // disregard emulation mode as unusable for our purposes
  for (i=0; i<devcount; i++) {
    int devmajor, devminor, rc;

    rc = vmd_cuda_device_props(i, NULL, 0, &devmajor, &devminor, 
                               NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL);

    if (rc == VMDCUDA_ERR_NONE) {
      // Check for emulation mode devices, and ignore if found
      if (((devmajor >= 1) && (devminor >= 0)) &&
          ((devmajor != 9999) && (devminor != 9999))) {
        usabledevs++;
      }
    } else if (rc != VMDCUDA_ERR_EMUDEVICE) {
      return VMDCUDA_ERR_SOMEDEVICES;
    }
  } 

  *numdev = usabledevs;

  return VMDCUDA_ERR_NONE;
}


void * vmd_cuda_devpool_setdevice(void * voidparms) {
  int count, id, dev;
  hipDeviceProp_t deviceProp;
  char *mesg;
  char *d_membuf;
  hipError_t err;

  wkf_threadpool_worker_getid(voidparms, &id, &count);
  wkf_threadpool_worker_getdata(voidparms, (void **) &mesg);
  wkf_threadpool_worker_getdevid(voidparms, &dev);

  /* set active device */
  hipSetDevice(dev);

#if CUDART_VERSION >= 2000
  /* Query SM count and clock rate, and compute a speed scaling value */
  /* the current code uses a GeForce GTX 280 / Tesla C1060 as the     */
  /* "1.0" reference value, with 30 SMs, and a 1.3 GHz clock rate     */ 
  memset(&deviceProp, 0, sizeof(hipDeviceProp_t));
  if (hipGetDeviceProperties(&deviceProp, dev) == hipSuccess) {
    float smscale = ((float) deviceProp.multiProcessorCount) / 30.0f;
    double clockscale = ((double) deviceProp.clockRate) / 1295000.0;
    float speedscale = smscale * ((float) clockscale);

#if 0
    printf("clock rate: %lf\n", (double) deviceProp.clockRate);
    printf("scale: %.4f smscale: %.4f clockscale: %.4f\n", 
           speedscale, smscale, clockscale);  
#endif

#if CUDART_VERSION >= 2030
    if (deviceProp.canMapHostMemory != 0) {
#if 0
      printf("Enabled mapped host memory on device[%d]\n", dev);
#endif

      /* 
       * set blocking/yielding API behavior and enable mapped host memory
       * If this fails, then either we've got a problematic device, or 
       * we have already set the device flags within this thread (shouldn't
       * ever happen), or the device we're accessing doesn't actually support
       * mapped host memory (shouldn't ever happen since we check for that).
       */

#if defined(VMDLIBOPTIX)
      // when compiled with OptiX enabled, we tell the CUDA runtime to 
      // maintain the peak local memory size that occured at runtime
      // to avoid thrashing with difficult scenes
      err = hipSetDeviceFlags(hipDeviceScheduleAuto | hipDeviceMapHost | hipDeviceLmemResizeToMax);
#else
      err = hipSetDeviceFlags(hipDeviceScheduleAuto | hipDeviceMapHost);
#endif
      if (err != hipSuccess) {
        printf("Warning) thread[%d] can't set GPU[%d] device flags\n", id, dev);
        printf("Warning) CUDA error: %s\n", hipGetErrorString(err)); 
      }
    }
#endif

    wkf_threadpool_worker_setdevspeed(voidparms, speedscale);

    /* 
     * Do a small 1MB device memory allocation to ensure that our context
     * has actually been initialized by the time we return.
     * If this tiny allocation fails, then something is seriously wrong
     * and we should mark this device as unusable for the rest of 
     * this VMD session.
     */
    if ((err = hipMalloc((void **) &d_membuf, 1*1024*1024)) == hipSuccess) {
      hipFree(d_membuf); 
    } else {
      printf("Warning) thread[%d] can't init GPU[%d] found by device query\n", id, dev); 
      printf("Warning) CUDA error: %s\n", hipGetErrorString(err));
      /* 
       * XXX we should mark the device unusable here so that no other code
       *     touchies it, but have no mechanism for doing that yet...
       */
    }
  }
#endif

  if (mesg != NULL)
    printf("devpool thread[%d / %d], device %d message: '%s'\n", id, count, dev, mesg);

  return NULL;
}



#if defined(__cplusplus)
}
#endif


