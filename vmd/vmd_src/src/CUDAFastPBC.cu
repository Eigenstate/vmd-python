#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 1995-2019 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAFastPBC.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.2 $       $Date: 2019/01/17 21:20:58 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *   Fast PBC wrapping code.
 ***************************************************************************/
#include <stdio.h>

// XXX this needs to go
#include "sys/time.h"

// Uses thrust for vector ops, various scan() reductions, etc.
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/device_vector.h>

#include "FastPBC.h"

/// XXX globals need to go...
const int nStreams = 4;
const int threads = 128;

__global__ void inverseboxsize (float *boxsize, float* invboxsize) {
	int tid = threadIdx.x;
	if (tid < 3) {
		invboxsize[tid] = 1.0 / boxsize[tid];
	}
}


// This is an inefficient kernel. Much slower than the one that replaced 
// it below. (~100 us)
__global__ void repositionfragments(int fnum, float *pos, int *compoundmap, 
                                    int *indexlist, float *boxsize, 
                                    float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k;
	for (int l = tid; l < fnum; l+=blockDim.x * gridDim.x) {
		float ccenter[3];
		int lowbound = compoundmap[l];
		int highbound = compoundmap[l+1];
		i = indexlist[lowbound];
		//Use the first element within the compound as the center.
		for (j=0; j < 3; j++) {
			ccenter[j] = pos[i*3+j];
		}
		//move the compound, wrapping it to be within half a box dimension from the center
		for (k = lowbound; k < highbound; k++ ) {
			i = indexlist[k];
			for (j=0; j < 3; j++) {
				pos[i*3+j] = pos[i*3+j] - (rintf((pos[i*3+j] - ccenter[j]) * invboxsize[j]) * boxsize[j]);
			}
		}
	}
}


// Super-efficient kernel. ~8 us execution time
__global__ void repositionfragments(float *pos, int sellen, int *atomtofragmap,
                                    int *compoundmap, int *indexlist, 
                                    float *boxsize, float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int cidx = indexlist[compoundmap[atomtofragmap[idx]]];
		//printf("tid: %d, idx %d, dim %d, cidx %d\n", tid, idx, dim, cidx);
		float center = pos[3 * cidx + dim];
		pos[3*indexlist[idx]+dim] = pos[3*indexlist[idx]+dim] - (rintf((pos[3*indexlist[idx]+dim] - center) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void wrapcompound(float *pos, int sellen, float *center, 
                             int *atomtofragmap, int *indexlist, 
                             float *boxsize, float *invboxsize, 
                             float *fragcenters) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int frag = atomtofragmap[idx];
		int aidx = indexlist[idx];
		pos[3*aidx+dim] = pos[3*aidx+dim] - (rintf((fragcenters[3*frag+dim] - center[dim]) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void wrapatomic(float *pos, int sellen, float *center, 
                           int *indexlist, float *boxsize, float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int aidx = indexlist[idx];
		pos[3*aidx+dim] = pos[3*aidx+dim] - (rintf((pos[3*aidx+dim] - center[dim]) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void unwrapatomic(float *pos, float *prev, int sellen, 
                             int *indexlist, 
                             float *boxsize, float *invboxsize) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < 3*sellen) {
		int idx = tid / 3;
		int dim = tid % 3;
		int aidx = indexlist[idx];
		pos[3*aidx+dim] = pos[3*aidx+dim] - (rintf((pos[3*aidx+dim] - prev[3*aidx+dim]) * invboxsize[dim]) * boxsize[dim]);
	}
}


__global__ void fragmentperatom(int fnum, int *compoundmap, 
                                int *atomtofragmap) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < fnum && tid != 0) {
		atomtofragmap[compoundmap[tid]] = 1;
	}
}


// XXX this likely duplicates stuff from prototypes in CUDAMeasure
__global__ void measurecenter(float *pos, float *center, int len, 
                              float *weights, int *weightidx, float *wscale) {
	__shared__ float reduce[96]; //96 is not an arbitrary number. Its divisible by 3! This lets us use
	//aligned memory accesses.
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	float mcenter = 0;
	int dim;
	if (tid < 3*len) {
		int idx = tid / 3;
		dim = tid % 3;
		mcenter = pos[3 * weightidx[idx] + dim] * weights[idx] * (*wscale);
	}
	reduce[threadIdx.x] = mcenter;
	__syncthreads();
	if (threadIdx.x < 48) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 48];
	}
	__syncthreads();
	if (threadIdx.x < 24) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 24];
	}
	__syncthreads();
	if (threadIdx.x < 12) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 12];
	}
	__syncthreads();
	if (threadIdx.x < 3) {
		mcenter = reduce[threadIdx.x] + reduce[threadIdx.x + 3] + reduce[threadIdx.x + 6] + reduce[threadIdx.x + 9];
		atomicAdd(&center[dim], mcenter);
	}
}


// Only differs from measurecenter based on how the weights are indexed.
// Here the expectation is that the full mass array has been passed, 
// so we need to find only specific elements of the weight array.
__global__ void measurecenter_fullmass(float *pos, float *center, int len, 
                                       float *weights, int *weightidx, 
                                       float *wscale) {
	__shared__ float reduce[96]; //96 is not an arbitrary number. Its divisible by 3! This lets us use
	//aligned memory accesses.
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	float mcenter = 0;
	int dim;
	if (tid < 3*len) {
		int idx = tid / 3;
		dim = tid % 3;
		int widx = weightidx[idx];
		mcenter = pos[3 * widx + dim] * weights[widx] * (*wscale);
	}
	reduce[threadIdx.x] = mcenter;
	__syncthreads();
	if (threadIdx.x < 48) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 48];
	}
	__syncthreads();
	if (threadIdx.x < 24) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 24];
	}
	__syncthreads();
	if (threadIdx.x < 12) {
		reduce[threadIdx.x] += reduce[threadIdx.x + 12];
	}
	__syncthreads();
	if (threadIdx.x < 3) {
		mcenter = reduce[threadIdx.x] + reduce[threadIdx.x + 3] + reduce[threadIdx.x + 6] + reduce[threadIdx.x + 9];
		atomicAdd(&center[dim], mcenter);
	}
}


// Harrumph. This kernel is inefficient. Less inefficient than the prettier 
// way of doing it, but this has only 1 kernel call, whereas the other one 
// had as many calls as there were fragments.
__global__ void computefragcenters(float *pos, float *centers, int fnum, 
                                   float *weights, float *wscale, 
                                   int *compoundmap, int *indexlist) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int i, j, k, f;
	float ccenter = 0;
	if (tid < 3*fnum) {
		f = tid / 3;
		j = tid % 3;
		int lowbound = compoundmap[f];
		int highbound = compoundmap[f+1];
		//Find the center of the compound.
		for (k = lowbound; k < highbound; k++ ) {
			i = indexlist[k];
			ccenter += pos[i*3+j] * weights[i] * wscale[f];
		}
		centers[3*f+j] = ccenter ;
	}
}


__global__ void fragwscale(float *fragscales, float *prefixsums, int fragnum, 
                           int *compoundmap) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < fragnum) {
		fragscales[tid] = 1.0 / (prefixsums[compoundmap[tid+1]] - prefixsums[compoundmap[tid]]);
	}
}


// XXX this is redundant in normal VMD builds, so we'll chop it later...
class Timer {
public:
	timeval t;
	Timer() {
		gettimeofday(&t, NULL);
	}
	~Timer() {
		timeval theend;
		gettimeofday(&theend,NULL);
		double elapsedTime;
		elapsedTime = (theend.tv_sec - t.tv_sec) * 1000000.0;  //sec to us
		elapsedTime += (theend.tv_usec - t.tv_usec);
		printf("Time: %.f us\n", elapsedTime);
	}
};


void fpbc_exec_unwrap(Molecule* mol, int first, int last, int sellen, int* indexlist) {
	Timestep *ts;
	int f;
	float *pos;
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpuinvboxsize[nStreams];
	int *gpuindexlist;
	hipStream_t stream[nStreams];
	hipEvent_t events[nStreams];
	int blocks = (3*sellen + threads - 1) / threads;
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipEventCreateWithFlags(&events[f], hipEventDisableTiming);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
	}
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipHostMalloc(&pos, nStreams * sizeof(float) * 3*mol->nAtoms);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	f = first;
	ts = mol->get_frame(f);
	memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
	hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
	hipEventRecord(events[f%nStreams],stream[f%nStreams]);
	//Do stuff
	for (f=first+1; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		if (! (f-(first+1) < nStreams)) {
			memcpy(mol->get_frame(f-nStreams)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
		}
		memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
		hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float) * 3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		//We must wait until the previous stream is done moving atoms around or loading. This part is inherently serial.
		hipStreamWaitEvent(stream[f%nStreams], events[(f-1)%nStreams],0);
		unwrapatomic<<<blocks,threads,0,stream[f%nStreams]>>>(gpupos[f%nStreams], gpupos[(f-1)%nStreams], sellen, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		hipEventRecord(events[f%nStreams],stream[f%nStreams]);
		//Copy out.
		hipMemcpyAsync(&pos[(f%nStreams)*3*mol->nAtoms], gpupos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToHost,stream[f%nStreams]);
	}
	//Copy back the remaining elements.
	for (f = max(last - nStreams + 1,0); f <= last; f++) {
		hipStreamSynchronize(stream[f%nStreams]);
		memcpy(mol->get_frame(f)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
	}
	//Cleanup
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipEventDestroy(events[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
	}
	hipHostUnregister(indexlist);
	hipHostUnregister(boxsize);
	hipFree(gpuindexlist);
	hipHostFree(pos);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_unwrap_cpu(mol, first, last, sellen, indexlist);
	}
}


void fpbc_exec_wrapcompound(Molecule* mol, int first, int last, int fnum, int *compoundmap, int sellen, int* indexlist, float* weights, AtomSel* csel, float* center, float* massarr) {
	//Declare variables.
	int f, i, j;
	Timestep *ts;
	float *pos;
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpucenters[nStreams];
	float *gpuinvboxsize[nStreams];
	float *gpuweights;
	float *wscale;
	float *gpufragweight;
	float *gpuwscan;
	int *gpuweightidx;
	int *gpuindexlist;
	int *gpuatomtofragmap;
	int *gpucompoundmap;
	float *gpufragcenters[nStreams];
	int blocks_frag = (fnum + threads - 1) / threads;
	hipStream_t stream[nStreams];
	//Allocate memory for static things (weight sums, maps, etc.)
	hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
	hipMalloc((void**) &wscale, sizeof(float));
	hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
	hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipMalloc((void**) &gpuweights, sizeof(float) * mol->nAtoms);
	hipMalloc((void**) &gpufragweight, sizeof(float) * fnum);
	hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipMemcpy(gpuweights, massarr, sizeof(float) * mol->nAtoms, hipMemcpyHostToDevice);//Unlike in wrapatomic, we'll pass over the full mass array to the GPU, since odds are we'll need it.
	hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	//Make the atomtofragmap by setting elements to 1 and then doing a scan.
	fragmentperatom<<<blocks_frag, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
	thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
	//Get the mass per fragment (for scaling/finding center of mass for everything.)
	thrust::device_vector<float> mass (thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights+mol->nAtoms));
	mass.push_back(0.0);//This zero is here so we can scan the weights efficiently.
	hipMalloc((void**) &gpuwscan, sizeof(float) * (mol->nAtoms+1));
	thrust::exclusive_scan(mass.begin(), mass.end(), thrust::device_ptr<float>(gpuwscan), 0, thrust::plus<float>());
	fragwscale<<<blocks_frag, threads>>>(gpufragweight, gpuwscan, fnum, gpucompoundmap);
	hipHostRegister(boxsize, sizeof(float)*3,0);
	hipHostMalloc(&pos, nStreams * sizeof(float) * 3*mol->nAtoms);
	if (csel != NULL) {
		hipMalloc((void**) &gpuweightidx, sizeof(int) * csel->selected);
		int *weightidx = new int[csel->selected];
		j=0;
		for (i=csel->firstsel; i<=csel->lastsel; i++) {
			if (csel->on[i]) {
				weightidx[j++] = i;
			}
		}
		hipMemcpy(gpuweightidx, weightidx, sizeof(int) * csel->selected, hipMemcpyHostToDevice);
		thrust::device_vector<int> ids (thrust::device_ptr<int>(gpuweightidx), thrust::device_ptr<int>(gpuweightidx+csel->selected));
		float tmp = 1.0 / thrust::reduce(thrust::make_permutation_iterator(mass.begin(), ids.begin()),
			thrust::make_permutation_iterator(mass.end(), ids.end()), 0, thrust::plus<float>());
		hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);
		delete [] weightidx;
	}
	//Allocate memory and create streams for per-frame changables.
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpucenters[f], sizeof(float) * 3);
		hipMemcpyAsync(gpucenters[f], center, sizeof(float)*3, hipMemcpyHostToDevice, stream[f]);
		hipMalloc((void**) &gpufragcenters[f], sizeof(float) * 3 * fnum);
	}
	hipDeviceSynchronize();
	hipFree(gpuwscan);
	//Start looping over the frames.
	int blocks = (3*sellen + threads - 1) / threads;
	blocks_frag = (3*fnum + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		if (! (f-first < nStreams)) {
			memcpy(mol->get_frame(f-nStreams)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
		}
		memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
		hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		if (csel != NULL) {
			hipMemsetAsync(gpucenters[f%nStreams],0, 3 * sizeof(float), stream[f%nStreams]);
			//Measure the center of the selection if one is provided. Put it into the 3-vector gpucenters.
			//To exploit some of the symmetry of the problem, pick a blocksize that is a multiple of 3, and preferably
			//also a multiple of the warpsize (96 is good!)
			measurecenter_fullmass<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
		}
		//Fragment centers need to be determined.
		//TODO: make this not suck. At the moment, I think this is the biggest bottleneck.
		computefragcenters<<<blocks_frag,threads,0,stream[f%nStreams]>>>(gpupos[f%nStreams], gpufragcenters[f%nStreams], fnum, gpuweights, gpufragweight, gpucompoundmap, gpuindexlist);
		//Wrap.
		wrapcompound<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuatomtofragmap, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams], gpufragcenters[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(&pos[(f%nStreams)*3*mol->nAtoms], gpupos[f%nStreams], sizeof(float) * 3 *mol->nAtoms, hipMemcpyDeviceToHost, stream[f%nStreams]);
	}
	//Cleanup
	//Copy back the remaining elements.
	for (f = max(last - nStreams + 1,0); f <= last; f++) {
		hipStreamSynchronize(stream[f%nStreams]);
		memcpy(mol->get_frame(f)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
	}
	//Free memory.
	hipHostUnregister(boxsize);
	hipHostUnregister(compoundmap);
	hipHostUnregister(indexlist);
	hipFree(gpucompoundmap);
	hipFree(gpuindexlist);
	hipFree(gpuatomtofragmap);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
	}
	hipHostFree(pos);
	hipFree(wscale);
	hipFree(gpufragweight);
	hipFree(gpuweights);
	if (csel != NULL) {
		hipFree(gpuweightidx);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_wrapcompound_cpu(mol, first, last, fnum, compoundmap, sellen, indexlist, weights, csel, center, massarr);
	}
}


void fpbc_exec_wrapatomic(Molecule* mol, int first, int last, int sellen, int* indexlist, 
	float* weights, AtomSel* csel, float* center) {
	int f, i, j;
	Timestep *ts;
	float *pos;
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpucenters[nStreams];
	float *gpuinvboxsize[nStreams];
	float *gpuweights;
	float *wscale;
	hipMalloc((void**) &wscale, sizeof(float));
	int *gpuweightidx;
	int *gpuindexlist;
	//Prepare GPU memory and streams
	hipStream_t stream[nStreams];
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	hipHostRegister(center, sizeof(float) * 3,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);
	hipHostMalloc(&pos, nStreams * sizeof(float) * 3*mol->nAtoms);
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpucenters[f], sizeof(float) * 3);
		hipMemcpyAsync(gpucenters[f], center, sizeof(float)*3, hipMemcpyHostToDevice, stream[f]);
	}
	if (csel != NULL) {
		hipMalloc((void**) &gpuweights, sizeof(float) * csel->selected);
		hipMalloc((void**) &gpuweightidx, sizeof(int) * csel->selected);
		hipMemcpy(gpuweights, weights, sizeof(float) * csel->selected, hipMemcpyHostToDevice);
		int *weightidx = new int[csel->selected];
		j=0;
		for (i=csel->firstsel; i<=csel->lastsel; i++) {
			if (csel->on[i]) {
				weightidx[j++] = i;
			}
		}
		hipMemcpy(gpuweightidx, weightidx, sizeof(int) * csel->selected, hipMemcpyHostToDevice);
		float tmp = 1.0 / thrust::reduce(thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights + csel->selected), 0, thrust::plus<float>());
		hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);
		delete [] weightidx;
	}
	int blocks = (3*sellen + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		if (! (f-first < nStreams)) {
			memcpy(mol->get_frame(f-nStreams)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
		}
		memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
		hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		if (csel != NULL) {
			hipMemsetAsync(gpucenters[f%nStreams],0, 3 * sizeof(float), stream[f%nStreams]);
			//Measure the center of the selection if one is provided. Put it into the 3-vector gpucenters.
			//To exploit some of the symmetry of the problem, pick a blocksize that is a multiple of 3, and preferably
			//also a multiple of the warpsize (96 is good!)
			measurecenter<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
		}
		//Wrap.
		wrapatomic<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(&pos[(f%nStreams)*3*mol->nAtoms], gpupos[f%nStreams], sizeof(float) * 3 *mol->nAtoms, hipMemcpyDeviceToHost, stream[f%nStreams]);
	}
	//Cleanup. Wait for the kernels to complete.
	for (f = max(last - nStreams + 1,0); f <= last; f++) {
		hipStreamSynchronize(stream[f%nStreams]);
		memcpy(mol->get_frame(f)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
	}
	hipHostUnregister(boxsize);
	hipHostUnregister(center);
	hipFree(gpuindexlist);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
		hipFree(gpucenters[f]);
	}
	hipHostFree(pos);
	if (csel != NULL) {
		hipFree(gpuweights);
		hipFree(gpuweightidx);
	}
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_wrapatomic_cpu(mol, first, last, sellen, indexlist, weights, csel, center);
	}
}


void fpbc_exec_join(Molecule* mol, int first, int last, int fnum, int *compoundmap, int sellen, int* indexlist) {
	int f;
	float *pos;
	Timestep *ts;
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpuinvboxsize[nStreams];
	int *gpucompoundmap;
	int *gpuatomtofragmap;
	int *gpuindexlist;
	int blocks = (fnum + threads - 1) / threads;
	hipStream_t stream[nStreams];
	hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
	hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
	hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	fragmentperatom<<<blocks, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
	thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
	}
	hipHostRegister(boxsize, sizeof(float)*3,0);
	hipHostMalloc(&pos, nStreams * sizeof(float) * 3*mol->nAtoms);
	//Make sure the gpuatomtofragmap is set before proceeding.
	hipDeviceSynchronize();
	blocks = (3*sellen + threads - 1) / threads;
	for (f = first; f <= last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		if (! (f-first < nStreams)) {
			memcpy(mol->get_frame(f-nStreams)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
		}
		memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
		hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float)*3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		repositionfragments<<<blocks,threads, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], sellen, gpuatomtofragmap,
			gpucompoundmap, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		//Copy back.
		hipMemcpyAsync(&pos[(f%nStreams)*3*mol->nAtoms], gpupos[f%nStreams], sizeof(float) * 3 *mol->nAtoms, hipMemcpyDeviceToHost, stream[f%nStreams]);
	}
	for (f = max(last - nStreams + 1,0); f <= last; f++) {
		hipStreamSynchronize(stream[f%nStreams]);
		memcpy(mol->get_frame(f)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
	}
	hipHostUnregister(boxsize);
	hipHostUnregister(indexlist);
	hipHostUnregister(compoundmap);
	hipFree(gpucompoundmap);
	hipFree(gpuindexlist);
	hipFree(gpuatomtofragmap);
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
	}
	hipHostFree(pos);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_join(mol, first, last, fnum, compoundmap, sellen, indexlist);
	}
}


void fpbc_exec_recenter(Molecule* mol, int first, int last, int csellen, int* cindexlist, int fnum, int *compoundmap, int sellen, int* indexlist, float* weights, AtomSel* csel, float* massarr) {
	//The basic idea here is to pass the data back and forth only once while both unwrapping and rewrapping the trajectory.
	Timestep *ts;
	int f;
	float *pos;
	float *gpupos[nStreams];
	float boxsize[3];
	float *gpuboxsize[nStreams];
	float *gpucenters[nStreams];
	float *gpuinvboxsize[nStreams];
	float *gpufragcenters[nStreams];
	float *wscale;
	hipMalloc((void**) &wscale, sizeof(float));
	float *gpuweights;
	int *gpuweightidx;
	int *gpuindexlist;
	float *gpufragweight;
	float *gpuwscan;
	int *gpuatomtofragmap;
	int *gpucompoundmap;
	hipStream_t stream[nStreams];
	hipEvent_t events[nStreams];
	hipHostRegister(indexlist, sizeof(int) * sellen,0);
	hipHostRegister(cindexlist, sizeof(int) * csellen,0);
	hipHostRegister(boxsize, sizeof(float) * 3,0);
	int blocks = (3*sellen + threads - 1) / threads;
	int blocks_frag = (fnum + threads - 1) / threads;
	for (f = 0; f< nStreams; f++) {
		hipStreamCreate(&stream[f]);
		hipEventCreateWithFlags(&events[f], hipEventDisableTiming);
		hipMalloc((void**) &gpupos[f], sizeof(float) * 3*mol->nAtoms);
		hipMalloc((void**) &gpuboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpuinvboxsize[f], sizeof(float) * 3);
		hipMalloc((void**) &gpucenters[f], sizeof(float) * 3);
		if (fnum) {
			hipMalloc((void**) &gpufragcenters[f], sizeof(float) * 3 * fnum);
		}
	}
	float tmp;
	hipMalloc((void**) &gpuindexlist, sizeof(int) * sellen);
	hipHostMalloc(&pos, nStreams * sizeof(float) * 3*mol->nAtoms);
	hipMemcpy(gpuindexlist, indexlist, sizeof(int) * sellen, hipMemcpyHostToDevice);
	//Deal with computing the weighted center of mass.
	hipMalloc((void**) &gpuweightidx, sizeof(int) * csellen);
	hipMemcpy(gpuweightidx, cindexlist, sizeof(int) * csellen, hipMemcpyHostToDevice);
	if (fnum) {//Compound runs only.
		hipMalloc((void**) &gpuweights, sizeof(float) * mol->nAtoms);
		hipMalloc((void**) &gpufragweight, sizeof(float) * fnum);
		hipHostRegister(compoundmap, sizeof(int) * (fnum+1),0);
		hipMemcpy(gpuweights, massarr, sizeof(float) * mol->nAtoms, hipMemcpyHostToDevice);
		hipMalloc((void**) &gpucompoundmap, sizeof(int) * (fnum+1));
		hipMemcpy(gpucompoundmap, compoundmap, sizeof(int) * (fnum+1), hipMemcpyHostToDevice);
		hipMalloc((void**) &gpuatomtofragmap, sizeof(int) * sellen);
		hipMemset(gpuatomtofragmap, 0, sizeof(int) * sellen);
		fragmentperatom<<<blocks_frag, threads>>>(fnum, gpucompoundmap, gpuatomtofragmap);//Setup the gpu per atom map.
		thrust::inclusive_scan(thrust::device_ptr<int>(gpuatomtofragmap), thrust::device_ptr<int>(gpuatomtofragmap + sellen), thrust::device_ptr<int>(gpuatomtofragmap));
		//Get the mass per fragment (for scaling/finding center of mass for everything.)
		thrust::device_vector<float> mass (thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights+mol->nAtoms));
		mass.push_back(0);//This zero is here so we can scan the weights efficiently.
		hipMalloc((void**) &gpuwscan, sizeof(float) * (mol->nAtoms+1));
		thrust::exclusive_scan(mass.begin(), mass.end(), thrust::device_ptr<float>(gpuwscan), 0, thrust::plus<float>());
		fragwscale<<<blocks_frag, threads>>>(gpufragweight, gpuwscan, fnum, gpucompoundmap);
		thrust::device_vector<int> ids (thrust::device_ptr<int>(gpuweightidx), thrust::device_ptr<int>(gpuweightidx+csel->selected));
		tmp = 1.0 / thrust::reduce(thrust::make_permutation_iterator(mass.begin(), ids.begin()),
			thrust::make_permutation_iterator(mass.end(), ids.end()), 0, thrust::plus<float>());
		hipDeviceSynchronize();
		hipFree(gpuwscan);
	}
	else {
		hipMalloc((void**) &gpuweights, sizeof(float) * csel->selected);
		hipMemcpy(gpuweights, weights, sizeof(float) * csel->selected, hipMemcpyHostToDevice);
		tmp = 1.0 / thrust::reduce(thrust::device_ptr<float>(gpuweights), thrust::device_ptr<float>(gpuweights + csel->selected), 0, thrust::plus<float>());
	}
	hipMemcpy(wscale, &tmp, sizeof(float), hipMemcpyHostToDevice);
	//Setup the initial memcopies.
	f = first;
	ts = mol->get_frame(f);
	memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
	hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
	//Do stuff
	blocks_frag = (3*fnum + threads - 1) / threads;
	for (f=first; f<=last; f++) {
		ts = mol->get_frame(f);
		boxsize[0] = ts->a_length;
		boxsize[1] = ts->b_length;
		boxsize[2] = ts->c_length;
		//Block here just so that I don't overwrite a buffer.
		hipStreamSynchronize(stream[f%nStreams]);
		if (! (f-first < nStreams)) {
			memcpy(mol->get_frame(f-nStreams)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
		}
		hipMemcpyAsync(gpuboxsize[f%nStreams], boxsize, sizeof(float) * 3, hipMemcpyHostToDevice, stream[f%nStreams]);
		//Do math here.
		inverseboxsize<<<1,4,0,stream[f%nStreams]>>>(gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		if (f > first) {//These are the ones that also need to be unwrapped.
			memcpy(&pos[(f%nStreams)*3*mol->nAtoms], ts->pos, sizeof(float) * 3*mol->nAtoms);
			hipMemcpyAsync(gpupos[f%nStreams], &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms, hipMemcpyHostToDevice, stream[f%nStreams]);
			//We must wait until the previous stream is done moving atoms around or loading. This part is inherently serial.
			hipStreamWaitEvent(stream[f%nStreams], events[(f-1)%nStreams],0);
			unwrapatomic<<<blocks,threads,0,stream[f%nStreams]>>>(gpupos[f%nStreams], gpupos[(f-1)%nStreams], csellen, gpuweightidx, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
			hipEventRecord(events[f%nStreams],stream[f%nStreams]);
		}
		hipMemsetAsync(gpucenters[f%nStreams],0, 3 * sizeof(float), stream[f%nStreams]);		
		//Compounding will have a non-zero fnum.
		if (fnum) {
			measurecenter_fullmass<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
			//Wrap.
			wrapcompound<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuatomtofragmap, gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams], gpufragcenters[f%nStreams]);
		}
		else {
			measurecenter<<<(3*csel->selected + 95) / 96, 96, 0, stream[f%nStreams]>>>(gpupos[f%nStreams], gpucenters[f%nStreams], csel->selected, gpuweights, gpuweightidx, wscale);
			//Wrap.
			wrapatomic<<<blocks, threads, 0, stream[f%nStreams]>>> (gpupos[f%nStreams], sellen, gpucenters[f%nStreams], gpuindexlist, gpuboxsize[f%nStreams], gpuinvboxsize[f%nStreams]);
		}
		hipEventRecord(events[f%nStreams],stream[f%nStreams]);
		//Copy out.
		hipMemcpyAsync(&pos[(f%nStreams)*3*mol->nAtoms], gpupos[f%nStreams], sizeof(float) * 3*mol->nAtoms, hipMemcpyDeviceToHost,stream[f%nStreams]);
	}
	//Copy back the remaining elements.
	for (f = max(last - nStreams + 1,0); f <= last; f++) {
		hipStreamSynchronize(stream[f%nStreams]);
		memcpy(mol->get_frame(f)->pos, &pos[(f%nStreams)*3*mol->nAtoms], sizeof(float) * 3*mol->nAtoms);
	}
	//Cleanup
	for (f = 0; f< nStreams; f++) {
		hipStreamDestroy(stream[f]);
		hipEventDestroy(events[f]);
		hipFree(gpupos[f]);
		hipFree(gpuboxsize[f]);
		hipFree(gpuinvboxsize[f]);
		hipFree(gpucenters[f]);
		if (fnum) {
			hipFree(gpufragcenters[f]);
		}
	}
	hipHostUnregister(indexlist);
	hipHostUnregister(cindexlist);
	hipHostUnregister(boxsize);
	hipFree(gpuindexlist);
	hipFree(gpuweightidx);
	hipFree(gpuweights);
	hipFree(wscale);
	if (fnum) {
		hipFree(gpucompoundmap);
		hipFree(gpuatomtofragmap);
		hipFree(gpufragweight);
		hipHostUnregister(compoundmap);
	}
	hipHostFree(pos);
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and fallback to CPU
		printf("CUDA error: %s\n", hipGetErrorString(error));
		printf("Reverting to CPU algorithm\n");
		fpbc_exec_recenter_cpu(mol, first, last, csellen, cindexlist, fnum, compoundmap, sellen, indexlist, weights, csel, massarr);
	}
}
