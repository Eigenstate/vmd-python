#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007-2009 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *      $RCSfile: CUDAClearDevice.cu,v $
 *      $Author: johns $        $Locker:  $             $State: Exp $
 *      $Revision: 1.14 $      $Date: 2014/08/20 17:15:50 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *   CUDA utility to clear all global and constant GPU memory areas to 
 *   known values.
 *
 ***************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "utilities.h"
#include "CUDAKernels.h"

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, %s line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
  return NULL; }}

// a full-sized 64-kB constant memory buffer to use to clear
// any existing device state
__constant__ static float constbuf[16384];

// maximum number of allocations to use to soak up all available RAM
#define MAXLOOPS 16

void * vmd_cuda_devpool_clear_device_mem(void * voidparms) {
  int id, count, dev;
  char *bufs[MAXLOOPS];
  size_t bufszs[MAXLOOPS];
  float zerobuf[16 * 1024];
  memset(zerobuf, 0, sizeof(zerobuf));
  memset(bufs, 0, MAXLOOPS * sizeof(sizeof(char *)));
  memset(bufszs, 0, MAXLOOPS * sizeof(sizeof(size_t)));

  wkf_threadpool_worker_getid(voidparms, &id, &count);
  wkf_threadpool_worker_getdevid(voidparms, &dev);

  // clear constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(constbuf), zerobuf, sizeof(zerobuf), 0);
  CUERR

#if 0
  // 
  // Allocate, clear, and deallocate all global memory we can touch
  //
  // XXX on platforms where the GPU shares DRAM with the CPU such as
  //     Tegra K1, the old memory clearing approach is problematic. 
  //     The CPU might implement VM paging, and it'll just end up 
  //     paging itself to death if we try and get all GPU memory.
  //     Given modern GPU drivers being better about clearing data between
  //     apps, it might be best to skip this step for now and either hope
  //     that old data isn't laying around in global GPU memory, or else
  //     take a very different approach that is more compatible with 
  //     systems like Tegra K1 that have a single memory system for both
  //     the CPU and the GPU.
  //
  // XXX In MPI enabled builds, we skip the global memory clearing step 
  //     since multiple VMD processes may end up being mapped to the
  //     same node, sharing the same set of GPUs.  A better way of handling
  //     this would be either to perform the memory clear only on one 
  //     MPI rank per physical node, or to distribute GPUs among 
  //     VMD processes so no sharing occurs.
  //
#if !defined(VMDMPI)
  int verbose=0;
  if (getenv("VMDCUDAVERBOSE") != NULL)
    verbose=1;

  size_t sz(1024 * 1024 * 1024); /* start with 1GB buffer size */
  int i, bufcnt=0;
  size_t totalsz=0;
  for (i=0; i<MAXLOOPS; i++) {
    // Allocate the largest buffer we can get. If we fail, we reduce request
    // size to half of the previous, and try again until we reach the minimum
    // request size threshold.
    hipError_t rc;
    while ((sz > (16 * 1024 * 1024)) && 
           ((rc=hipMalloc((void **) &bufs[i], sz)) != hipSuccess)) {
      hipGetLastError(); // reset error state
      sz >>= 1;
    }

    if (rc == hipSuccess) {
      bufszs[i] = sz;
      totalsz += sz; 
      bufcnt++;
      if (verbose)
        printf("devpool thread[%d / %d], dev %d buf[%d] size: %d\n", id, count, dev, i, sz);
    } else {
      bufs[i] = NULL;
      bufszs[i] = 0;
      if (verbose)
        printf("devpool thread[%d / %d], dev %d buf[%d] failed min allocation size: %d\n", id, count, dev, i, sz);

      // terminate allocation loop early
      break;
    } 
  }

  if (verbose)
    printf("devpool thread[%d / %d], dev %d allocated %d buffers\n", id, count, dev, bufcnt);

  for (i=0; i<bufcnt; i++) {
    if ((bufs[i] != NULL) && (bufszs[i] > 0)) {
      hipMemset(bufs[i], 0, bufszs[i]);
      hipFree(bufs[i]);
      bufs[i] = NULL;
      bufszs[i] = 0;
    }
  }
  CUERR

  if (verbose)
    printf("  Device %d cleared %d MB of GPU memory\n", dev, totalsz / (1024 * 1024));

#endif
#endif

  return NULL;
}

